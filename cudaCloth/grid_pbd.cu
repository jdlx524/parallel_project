#include "hip/hip_runtime.h"
// cloth_simulation.cpp

#include <GL/glew.h>
#include <GL/glut.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <iostream>
#include <cstring>
#include <vector>
#include <cmath>
#include <cstdlib>
#include <chrono>
#include <ctime>

#include <omp.h>

// 辅助宏
#define MAX(x,y) ((x) > (y) ? (x) : (y))
#define MIN(x,y) ((x) < (y) ? (x) : (y))
#define MAX_CAPACITY 32  // 每个网格单元最多存储 32 个顶点

// error 宏：用于检查 CUDA 错误
#define CUERR {                                                              \
    hipError_t err;                                                         \
    if ((err = hipGetLastError()) != hipSuccess) {                         \
       std::cout << "CUDA error: " << hipGetErrorString(err) << " : "       \
                 << __FILE__ << ", line " << __LINE__ << std::endl;          \
       exit(1);                                                              \
    }                                                                        \
}

// HIQ 表示高精度版本（可调整网格尺寸与迭代次数）
#define HIQ

#ifndef HIQ
    #define N (64)
    #define EPS (5E-3)
    #define E (0.05)
    #define BIAS (0.17)
    #define ITERS (128)
    #define G (1)
#else
    #define N (256)
    #define EPS (5E-3)
    #define E (0.05)
    #define BIAS (0.15)
    #define ITERS (400)
    #define G (1)
#endif

// 窗口尺寸
const unsigned int window_width  = 1280;
const unsigned int window_height = 720;

// OpenGL 缓冲对象
static GLuint ibo = 0;
static GLuint vbo = 0;
static GLuint cbo = 0;
static GLuint nbo = 0;

// 主机侧数据（每 3 个浮点数一顶点，速度 3 个浮点数，颜色 4 个浮点数）
static std::vector<float> vertices;
static std::vector<float> velocities;
static std::vector<float> colors;
static std::vector<float> normals;
static std::vector<int> indices;

// 设备侧数据：通过 CUDA OpenGL 互操作获得 VBO 指针
struct hipGraphicsResource *vbo_resource;
struct hipGraphicsResource *vbo_resource2;  // 用于法线的 VBO

// 设备侧临时缓冲区（GPU 版 relax 操作使用）
float3 *Temp;

// 设备侧存储速度数据（按顶点，每顶点 3 个 float，共连续内存）
static float* d_velocities = nullptr;

// 设备侧锁数组，每个顶点对应一个锁（用于自碰撞时同步）
static int* d_locks = nullptr;

// 约束参数（在 init_data 中根据顶点间距计算）
static float cnstr_two;
static float cnstr_dia;

// 帧计数器
static size_t frames = 0;

static std::chrono::high_resolution_clock::time_point start_time;

/**************************************************
 * 辅助函数（设备/主机）
 **************************************************/
// 本情境下不需要地形修正，返回 0
__host__ __device__ __forceinline__
float terrain_height(float x, float y) {
    return 0.0f;
}

// update_positions：仅根据当前速度以欧拉法积分更新位置（无重力）
__host__ __device__ __forceinline__
void update_positions(float3& pos, float3& vel, const float eps) {
    pos.x += eps * vel.x;
    pos.y += eps * vel.y;
    pos.z += eps * vel.z;
}

// adjust_positions：本情境下不做额外校正（保留空操作）
__host__ __device__ __forceinline__
void adjust_positions(float3 &pos) {
    // no-op
}

// relax_constraint：用于约束相邻顶点距离
__host__ __device__ __forceinline__
void relax_constraint(const float3 *Pos, float3 *Tmp,
                      const int l, const int m,
                      const float constraint, const float bias) {
    float3 delta;
    delta.x = Pos[l].x - Pos[m].x;
    delta.y = Pos[l].y - Pos[m].y;
    delta.z = Pos[l].z - Pos[m].z;
    const float invlen = rsqrtf(delta.x * delta.x + delta.y * delta.y + delta.z * delta.z);
    const float factor = (1.0f - constraint * invlen) * bias;
#if defined(__CUDA_ARCH__)
    atomicAdd(&Tmp[l].x, -delta.x * factor);
    atomicAdd(&Tmp[l].y, -delta.y * factor);
    atomicAdd(&Tmp[l].z, -delta.z * factor);
    atomicAdd(&Tmp[m].x, +delta.x * factor);
    atomicAdd(&Tmp[m].y, +delta.y * factor);
    atomicAdd(&Tmp[m].z, +delta.z * factor);
#else
    Tmp[l].x -= delta.x * factor;
    Tmp[l].y -= delta.y * factor;
    Tmp[l].z -= delta.z * factor;
    Tmp[m].x += delta.x * factor;
    Tmp[m].y += delta.y * factor;
    Tmp[m].z += delta.z * factor;
#endif
}

// normalize：将向量归一化
__host__ __device__ __forceinline__
void normalize(float3 &normal) {
    const float invrho = rsqrtf(normal.x*normal.x + normal.y*normal.y + normal.z*normal.z);
    normal.x *= invrho;
    normal.y *= invrho;
    normal.z *= invrho;
}

// wedge：计算局部面片法线贡献
__host__ __device__ __forceinline__
void wedge(const float3 *Vertices, float3 &Normal,
           const int &i, const int &j, const int n,
           const int &a, const int &b) {
    float3 center = Vertices[i*n+j];
    float3 span_u = Vertices[(i+a)*n+j];
    float3 span_v = Vertices[i*n+(j+b)];
    span_u.x -= center.x; span_u.y -= center.y; span_u.z -= center.z;
    span_v.x -= center.x; span_v.y -= center.y; span_v.z -= center.z;
    float3 cross;
    cross.x = span_u.y * span_v.z - span_v.y * span_u.z;
    cross.y = span_u.z * span_v.x - span_v.z * span_u.x;
    cross.z = span_u.x * span_v.y - span_v.x * span_u.y;
    Normal.x += cross.x * a * b;
    Normal.y += cross.y * a * b;
    Normal.z += cross.z * a * b;
}

/**************************************************
 * CUDA 内核函数
 **************************************************/

// 构造网格内核
__global__
void build_grid_kernel(const float* vertices, int total, 
    float cellSize, float xMin, float yMin, float zMin,
    int gridDimX, int gridDimY, int gridDimZ,
    int maxCapacity, int* cellCounts, int* cellIndices)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= total) return;
    
    // 读取顶点位置（假定每个顶点由连续的 3 个 float 表示）
    float x = vertices[idx*3];
    float y = vertices[idx*3 + 1];
    float z = vertices[idx*3 + 2];
    
    // 计算该顶点所在的格子下标（向下取整）
    int ix = (int)((x - xMin) / cellSize);
    int iy = (int)((y - yMin) / cellSize);
    int iz = (int)((z - zMin) / cellSize);
    // 边界检查
    if(ix < 0) ix = 0; else if(ix >= gridDimX) ix = gridDimX - 1;
    if(iy < 0) iy = 0; else if(iy >= gridDimY) iy = gridDimY - 1;
    if(iz < 0) iz = 0; else if(iz >= gridDimZ) iz = gridDimZ - 1;
    
    int cellIdx = ix + iy * gridDimX + iz * gridDimX * gridDimY;
    // 原子加：获得在 cellIdx 格子中的插入槽位
    int slot = atomicAdd(&cellCounts[cellIdx], 1);
    if(slot < maxCapacity) {
        cellIndices[cellIdx * maxCapacity + slot] = idx;
    }
}

// 传播内核：根据速度更新位置
__global__
void propagate_kernel(float3 *Vertices, float3 *Velocities, const int n, const float eps=EPS) {
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    if(i < n && j < n)
        update_positions(Vertices[i*n+j], Velocities[i*n+j], eps);
}

// 校正内核：根据约束迭代调整位置（调用 relax_constraint）
__global__
void validate_kernel(float3 *Vertices, float3 *Temp,
                     const float cnstr_two, const float cnstr_dia,
                     const int n, const float bias=BIAS) {
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    if(i < n-1 && j < n)
        relax_constraint(Vertices, Temp, i*n+j, (i+1)*n+j, cnstr_two, bias);
    if(i < n && j < n-1)
        relax_constraint(Vertices, Temp, i*n+j, i*n+(j+1), cnstr_two, bias);
    if(i < n-2 && j < n)
        relax_constraint(Vertices, Temp, i*n+j, (i+2)*n+j, 2*cnstr_two, bias);
    if(i < n && j < n-2)
        relax_constraint(Vertices, Temp, i*n+j, i*n+(j+2), 2*cnstr_two, bias);
    if(i < n-1 && j < n-1)
        relax_constraint(Vertices, Temp, i*n+j, (i+1)*n+(j+1), cnstr_dia, bias);
    if(i > 0 && i < n && j < n-1)
        relax_constraint(Vertices, Temp, i*n+j, (i-1)*n+(j+1), cnstr_dia, bias);
}

// 调整内核：调用 adjust_positions（目前为空操作）
__global__
void adjust_kernel(float3* Temp, const int n) {
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    if(i < n && j < n)
        adjust_positions(Temp[i*n+j]);
}

// 更新法线内核：根据顶点数据计算法线
__global__
void update_normals_kernel(float3 *Vertices, float3 *Normals, const int n) {
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    if(i < n && j < n) {
        float3 Normal = {0, 0, 0};
        if(i > 0 && j > 0)
            wedge(Vertices, Normal, i, j, n, -1, -1);
        if(i > 0 && j+1 < n)
            wedge(Vertices, Normal, i, j, n, -1, +1);
        if(i+1 < n && j > 0)
            wedge(Vertices, Normal, i, j, n, +1, -1);
        if(i+1 < n && j+1 < n)
            wedge(Vertices, Normal, i, j, n, +1, +1);
        normalize(Normal);
        Normals[i*n+j] = Normal;
    }
}

// 自碰撞内核（CUDA版）
// 每个线程针对一个顶点 i，遍历 j > i，检测碰撞，若碰撞则更新位置及消除相对速度
__global__
void self_collision_kernel(float* vertices, float* velocities,
                           int n, float collisionThreshold, int total, int* locks) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= total) return;
    
    int row_i = i / n, col_i = i % n;
    
    for (int j = i + 1; j < total; j++) {
        int row_j = j / n, col_j = j % n;
        // 排除局部邻域（相邻点不考虑）
        if (abs(row_i - row_j) <= 1 && abs(col_i - col_j) <= 1)
            continue;
        
        // 读取顶点位置（每个顶点 3 个 float）
        float xi = vertices[i*3];
        float yi = vertices[i*3 + 1];
        float zi = vertices[i*3 + 2];
        float xj = vertices[j*3];
        float yj = vertices[j*3 + 1];
        float zj = vertices[j*3 + 2];
        
        float dx = xi - xj;
        float dy = yi - yj;
        float dz = zi - zj;
        float dist2 = dx*dx + dy*dy + dz*dz;
        
        if(dist2 < collisionThreshold * collisionThreshold) {
            // 计算平均位置
            float avgx = 0.5f * (xi + xj);
            float avgy = 0.5f * (yi + yj);
            float avgz = 0.5f * (zi + zj);
            
            // 采用锁机制：先锁下标较小的，再锁较大的
            while (atomicCAS(&locks[i], 0, 1) != 0);
            while (atomicCAS(&locks[j], 0, 1) != 0);
            
            // 更新两个顶点位置（写回平均值）
            vertices[i*3]     = avgx;
            vertices[i*3 + 1] = avgy;
            vertices[i*3 + 2] = avgz;
            vertices[j*3]     = avgx;
            vertices[j*3 + 1] = avgy;
            vertices[j*3 + 2] = avgz;
            
            // 消除相对速度在碰撞方向上的分量
            float vix = velocities[i*3];
            float viy = velocities[i*3 + 1];
            float viz = velocities[i*3 + 2];
            float vjx = velocities[j*3];
            float vjy = velocities[j*3 + 1];
            float vjz = velocities[j*3 + 2];
            float rvx = vix - vjx;
            float rvy = viy - vjy;
            float rvz = viz - vjz;
            float norm = sqrtf(dx*dx + dy*dy + dz*dz);
            if(norm > 0){
                float nx = dx / norm;
                float ny = dy / norm;
                float nz = dz / norm;
                float vrel = rvx * nx + rvy * ny + rvz * nz;
                float correction = vrel / 2.0f;
                velocities[i*3]     -= correction * nx;
                velocities[i*3 + 1] -= correction * ny;
                velocities[i*3 + 2] -= correction * nz;
                velocities[j*3]     += correction * nx;
                velocities[j*3 + 1] += correction * ny;
                velocities[j*3 + 2] += correction * nz;
            }
            
            // 释放锁
            atomicExch(&locks[i], 0);
            atomicExch(&locks[j], 0);
        }
    }
}

// 自碰撞基于网格内核（CUDA版）
// 每个线程针对一个顶点，根据所在格子及周围 26 个格子中的候选顶点检测碰撞
__global__
void self_collision_grid_kernel(float* vertices, float* velocities,
       int total, float collisionThreshold, float cellSize, float xMin, float yMin, float zMin,
       int gridDimX, int gridDimY, int gridDimZ, int maxCapacity,
       const int* cellCounts, const int* cellIndices,
       int* locks)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= total) return;
    
    // 加载当前顶点 idx 的位置
    float xi = vertices[idx*3];
    float yi = vertices[idx*3 + 1];
    float zi = vertices[idx*3 + 2];
    // 计算当前顶点所在的网格坐标
    int ix = (int)((xi - xMin) / cellSize);
    int iy = (int)((yi - yMin) / cellSize);
    int iz = (int)((zi - zMin) / cellSize);
    if(ix < 0) ix = 0; else if(ix >= gridDimX) ix = gridDimX - 1;
    if(iy < 0) iy = 0; else if(iy >= gridDimY) iy = gridDimY - 1;
    if(iz < 0) iz = 0; else if(iz >= gridDimZ) iz = gridDimZ - 1;
    
    // 检查当前格子及其 26 个邻域格子
    for (int dz = -1; dz <= 1; dz++){
        int nz = iz + dz;
        if(nz < 0 || nz >= gridDimZ) continue;
        for (int dy = -1; dy <= 1; dy++){
            int ny = iy + dy;
            if(ny < 0 || ny >= gridDimY) continue;
            for (int dx = -1; dx <= 1; dx++){
                int nx = ix + dx;
                if(nx < 0 || nx >= gridDimX) continue;
                int cellIdx = nx + ny * gridDimX + nz * gridDimX * gridDimY;
                int count = cellCounts[cellIdx];
                // 遍历该格子内的所有顶点
                for (int k = 0; k < count && k < maxCapacity; k++){
                    int j = cellIndices[cellIdx * maxCapacity + k];
                    // 避免重复检测（只处理 j > idx 的顶点）
                    if(j <= idx) continue;
                    
                    // 读取候选顶点 j 的位置
                    float xj = vertices[j*3];
                    float yj = vertices[j*3 + 1];
                    float zj = vertices[j*3 + 2];
                    float dx_ = xi - xj;
                    float dy_ = yi - yj;
                    float dz_ = zi - zj;
                    float dist2 = dx_*dx_ + dy_*dy_ + dz_*dz_;
                    if(dist2 < collisionThreshold * collisionThreshold) {
                        // 如果检测到碰撞，计算两个顶点的平均位置
                        float avgx = 0.5f * (xi + xj);
                        float avgy = 0.5f * (yi + yj);
                        float avgz = 0.5f * (zi + zj);
                        // 采用原子锁机制更新数据
                        while (atomicCAS(&locks[idx], 0, 1) != 0);
                        while (atomicCAS(&locks[j], 0, 1) != 0);
                        
                        // 更新顶点位置：写入平均值
                        vertices[idx*3]     = avgx;
                        vertices[idx*3 + 1] = avgy;
                        vertices[idx*3 + 2] = avgz;
                        vertices[j*3]       = avgx;
                        vertices[j*3 + 1]   = avgy;
                        vertices[j*3 + 2]   = avgz;
                        
                        // 消除沿碰撞方向的相对速度
                        float vix = velocities[idx*3];
                        float viy = velocities[idx*3 + 1];
                        float viz = velocities[idx*3 + 2];
                        float vjx = velocities[j*3];
                        float vjy = velocities[j*3 + 1];
                        float vjz = velocities[j*3 + 2];
                        float rvx = vix - vjx;
                        float rvy = viy - vjy;
                        float rvz = viz - vjz;
                        float norm = sqrtf(dx_*dx_ + dy_*dy_ + dz_*dz_);
                        if(norm > 0){
                            float nx = dx_ / norm;
                            float ny = dy_ / norm;
                            float nz = dz_ / norm;
                            float vrel = rvx * nx + rvy * ny + rvz * nz;
                            float correction = vrel*(1+E) / 2.0f;
                            velocities[idx*3]     -= correction * nx;
                            velocities[idx*3 + 1] -= correction * ny;
                            velocities[idx*3 + 2] -= correction * nz;
                            velocities[j*3]       += correction * nx;
                            velocities[j*3 + 1]   += correction * ny;
                            velocities[j*3 + 2]   += correction * nz;
                        }
                        
                        // 释放锁
                        atomicExch(&locks[idx], 0);
                        atomicExch(&locks[j], 0);
                    }
                }
            }
        }
    }
}

/**************************************************
 * GPU 包装调用函数
 **************************************************/
 
// 复制内核（供 validate_gpu 使用）
__global__
void copy_kernel(float3 *target, float3 *source, int n) {
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    if(i < n && j < n)
        target[i*n+j] = source[i*n+j];
}

// propagate_gpu：利用 CUDA 内核更新顶点位置（欧拉积分）
void propagate_gpu(int n) {
    float3 *d_vertices;
    size_t num_bytes;
    hipGraphicsMapResources(1, &vbo_resource, 0);
    hipGraphicsResourceGetMappedPointer((void **)&d_vertices, &num_bytes, vbo_resource);
    
    dim3 grid((n+7)/8, (n+7)/8, 1), blck(8,8,1);
    propagate_kernel<<<grid, blck>>>(d_vertices, reinterpret_cast<float3*>(d_velocities), n);
    CUERR
    hipGraphicsUnmapResources(1, &vbo_resource, 0);
}

// validate_gpu：利用 CUDA 内核多次迭代校正顶点间距离
void validate_gpu(int n, const int iters=ITERS) {
    float3 *d_vertices;
    size_t num_bytes;
    hipGraphicsMapResources(1, &vbo_resource, 0);
    hipGraphicsResourceGetMappedPointer((void **)&d_vertices, &num_bytes, vbo_resource);
    
    dim3 grid((n+7)/8, (n+7)/8, 1), blck(8,8,1);
    for(int iter = 0; iter < iters; iter++){
        // 将当前顶点数据复制到临时缓冲区 Temp
        copy_kernel<<<grid, blck>>>(Temp, d_vertices, n);
        CUERR
        validate_kernel<<<grid, blck>>>(d_vertices, Temp, cnstr_two, cnstr_dia, n);
        CUERR
        adjust_kernel<<<grid, blck>>>(Temp, n);
        CUERR
        copy_kernel<<<grid, blck>>>(d_vertices, Temp, n);
        CUERR
    }
    hipGraphicsUnmapResources(1, &vbo_resource, 0);
}

// 更新法线 GPU 版本
void update_normals_gpu(int n) {
    float3 *d_vertices, *d_normals;
    size_t num_bytes;
    hipGraphicsMapResources(1, &vbo_resource, 0);
    hipGraphicsResourceGetMappedPointer((void **)&d_vertices, &num_bytes, vbo_resource);
    hipGraphicsMapResources(1, &vbo_resource2, 0);
    hipGraphicsResourceGetMappedPointer((void **)&d_normals, &num_bytes, vbo_resource2);
    
    dim3 grid((n+7)/8, (n+7)/8, 1), blck(8,8,1);
    update_normals_kernel<<<grid, blck>>>(d_vertices, d_normals, n);
    CUERR
    hipGraphicsUnmapResources(1, &vbo_resource, 0);
    hipGraphicsUnmapResources(1, &vbo_resource2, 0);
}

// 自碰撞 GPU 版本包装函数（暴力版本）
void self_collision_gpu(int n) {
    float* d_vertices;
    size_t num_bytes;
    hipGraphicsMapResources(1, &vbo_resource, 0);
    hipGraphicsResourceGetMappedPointer((void **)&d_vertices, &num_bytes, vbo_resource);

    int total = n * n;
    int blockSize = 256;
    int numBlocks = (total + blockSize - 1) / blockSize;
    
    for(int i=0;i<4;++i)
        self_collision_kernel<<<numBlocks, blockSize>>>(d_vertices, d_velocities, n, 0.7f * cnstr_two, total, d_locks);
    hipDeviceSynchronize();
    CUERR
    hipGraphicsUnmapResources(1, &vbo_resource, 0);
}

// 自碰撞 GPU 版本包装函数（基于空间网格）
void self_collision_gpu_spatial(int n) {
    // 将 VBO 中的顶点数据映射到设备内存中
    float* d_vertices;
    size_t num_bytes;
    hipGraphicsMapResources(1, &vbo_resource, 0);
    hipGraphicsResourceGetMappedPointer((void **)&d_vertices, &num_bytes, vbo_resource);

    int total = n * n;
    
    // 使用碰撞阈值计算网格尺寸（这里采用 cellSize = 2 * collisionThreshold）
    float collisionThreshold = 0.5f * cnstr_two; 
    float cellSize = 2.0f * collisionThreshold;
    
    // 固定模拟区域边界
    float xMin = -2.0f, yMin = -2.0f, zMin = -1.0f;
    float xMax =  2.0f, yMax =  2.0f, zMax =  1.0f;
    int gridDimX = (int)ceilf((xMax - xMin) / cellSize);
    int gridDimY = (int)ceilf((yMax - yMin) / cellSize);
    int gridDimZ = (int)ceilf((zMax - zMin) / cellSize);
    int gridSize = gridDimX * gridDimY * gridDimZ;

    // 分配网格数组（cellCounts 和 cellIndices）在设备上的内存
    int* d_cellCounts = nullptr;
    int* d_cellIndices = nullptr;
    hipMalloc(&d_cellCounts, gridSize * sizeof(int));
    hipMalloc(&d_cellIndices, gridSize * MAX_CAPACITY * sizeof(int));
    hipMemset(d_cellCounts, 0, gridSize * sizeof(int));

    // 构建网格：为每个顶点确定其所在的格子，并记录顶点索引
    int threadsPerBlock = 256;
    int blocks = (total + threadsPerBlock - 1) / threadsPerBlock;
    build_grid_kernel<<<blocks, threadsPerBlock>>>(d_vertices, total, cellSize, xMin, yMin, zMin,
         gridDimX, gridDimY, gridDimZ, MAX_CAPACITY, d_cellCounts, d_cellIndices);
    hipDeviceSynchronize();
    CUERR

    // 启动基于网格的自碰撞检测内核
    blocks = (total + threadsPerBlock - 1) / threadsPerBlock;
    self_collision_grid_kernel<<<blocks, threadsPerBlock>>>(d_vertices, d_velocities, total, collisionThreshold,
         cellSize, xMin, yMin, zMin, gridDimX, gridDimY, gridDimZ, MAX_CAPACITY, d_cellCounts, d_cellIndices, d_locks);
    hipDeviceSynchronize();
    CUERR

    // 释放用于构造网格的临时内存
    hipFree(d_cellCounts);
    hipFree(d_cellIndices);
    hipGraphicsUnmapResources(1, &vbo_resource, 0);
}

/**************************************************
 * CPU 版本辅助函数（供对比，非 GPU 仿真时使用）
 **************************************************/

void propagate(float3 *vertices, float3 *velocities, const int n, const float eps=EPS) {
    #pragma omp parallel for collapse(2)
    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            update_positions(vertices[i*n+j], velocities[i*n+j], eps);
        }
    }
}

void validate(float3 *vertices, const int n, const int iters=ITERS, const float bias=BIAS) {
    std::vector<float3> temp(3*n*n);
    for(int iter = 0; iter < iters; iter++){
        std::memcpy(temp.data(), vertices, sizeof(float3)*n*n);
        #pragma omp parallel for collapse(2)
        for(int i = 0; i < n-1; i++){
            for(int j = 0; j < n; j++){
                relax_constraint(vertices, temp.data(), i*n+j, (i+1)*n+j, cnstr_two, bias);
            }
        }
        #pragma omp parallel for collapse(2)
        for(int i = 0; i < n; i++){
            for(int j = 0; j < n-1; j++){
                relax_constraint(vertices, temp.data(), i*n+j, i*n+(j+1), cnstr_two, bias);
            }
        }
        #pragma omp parallel for collapse(2)
        for(int i = 0; i < n-2; i++){
            for(int j = 0; j < n; j++){
                relax_constraint(vertices, temp.data(), i*n+j, (i+2)*n+j, 2*cnstr_two, bias);
            }
        }
        #pragma omp parallel for collapse(2)
        for(int i = 0; i < n; i++){
            for(int j = 0; j < n-2; j++){
                relax_constraint(vertices, temp.data(), i*n+j, i*n+(j+2), 2*cnstr_two, bias);
            }
        }
        #pragma omp parallel for collapse(2)
        for(int i = 0; i < n-1; i++){
            for(int j = 0; j < n-1; j++){
                relax_constraint(vertices, temp.data(), i*n+j, (i+1)*n+(j+1), cnstr_dia, bias);
            }
        }
        #pragma omp parallel for collapse(2)
        for(int i = 1; i < n; i++){
            for(int j = 0; j < n-1; j++){
                relax_constraint(vertices, temp.data(), i*n+j, (i-1)*n+(j+1), cnstr_dia, bias);
            }
        }
        #pragma omp parallel for collapse(2)
        for(int i = 0; i < n; i++){
            for(int j = 0; j < n; j++){
                adjust_positions(temp[i*n+j]);
            }
        }
        std::memcpy(vertices, temp.data(), sizeof(float3)*n*n);
    }
}

void update_normals(float3 *vertices, float3 *normals, const int n) {
    std::memset(normals, 0, sizeof(float3)*n*n);
    #pragma omp parallel for collapse(2)
    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            float3 normal = {0, 0, 0};
            if(i > 0 && j > 0)
                wedge(vertices, normal, i, j, n, -1, -1);
            if(i > 0 && j+1 < n)
                wedge(vertices, normal, i, j, n, -1, +1);
            if(i+1 < n && j > 0)
                wedge(vertices, normal, i, j, n, +1, -1);
            if(i+1 < n && j+1 < n)
                wedge(vertices, normal, i, j, n, +1, +1);
            normalize(normal);
            normals[i*n+j] = normal;
        }
    }
}

/**************************************************
 * OpenGL 渲染函数：完整 display 函数（采用全部 GPU 版仿真）
 **************************************************/
void display() {
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    // 帧计数增加
    frames++;
    // GPU 版仿真更新：传播、约束校正、自碰撞、更新法线
    propagate_gpu(N);
    validate_gpu(N, ITERS);
    self_collision_gpu_spatial(N);
    update_normals_gpu(N);

    // 启用顶点、颜色、法线、索引数组的客户端状态
    glEnableClientState(GL_VERTEX_ARRAY);
    glEnableClientState(GL_COLOR_ARRAY);
    glEnableClientState(GL_NORMAL_ARRAY);
    glEnableClientState(GL_INDEX_ARRAY);

    // 绑定顶点数据（VBO 已被 GPU 内核更新）
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(3, GL_FLOAT, 0, NULL);

    // 绑定颜色数据
    glBindBuffer(GL_ARRAY_BUFFER, cbo);
    glColorPointer(4, GL_FLOAT, 0, NULL);

    // 绑定法线数据
    glBindBuffer(GL_ARRAY_BUFFER, nbo);
    glNormalPointer(GL_FLOAT, 0, NULL);

    // 绘制布料网格（采用三角带）
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, ibo);
    glDrawElements(GL_TRIANGLE_STRIP, indices.size(), GL_UNSIGNED_INT, NULL);

    // 关闭客户端状态
    glDisableClientState(GL_VERTEX_ARRAY);
    glDisableClientState(GL_COLOR_ARRAY);
    glDisableClientState(GL_NORMAL_ARRAY);
    glDisableClientState(GL_INDEX_ARRAY);

    // 每 500 帧输出一次渲染耗时
    if(frames % 500 == 0)
    {
        auto end_time = std::chrono::high_resolution_clock::now();
        double duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
        std::cout << "Rendering 500 frames took " << duration << " ms" << std::endl;
        // 可重置 start_time 或重新初始化布料数据等
        start_time = std::chrono::high_resolution_clock::now();
    }
    glutSwapBuffers();
    glutPostRedisplay();
}

/**************************************************
 * init_data: 初始化布料数据，并分配相关设备内存
 * 这里构造主视倒 V 形态，左半边（y 较小）运动速度较大
 **************************************************/
void init_data(int n) {
    // 布料在 x,y 坐标上的取值范围：[-2, +2]
    float offsetX = 0.0f;
    float offsetY = 0.0f;

    // 设置形态参数：baseline 为中间最高，amplitude 控制下降幅度
    float baseline = 0.8f;
    float amplitude = 0.3f;

    float min_x = -2.0f + offsetX;
    float max_x =  2.0f + offsetX;
    float min_y = -2.0f + offsetY;
    float max_y =  2.0f + offsetY;
    // 为主视效果，以 y 坐标构造倒 V 形态，以 y 中心作为最高处
    float center_y = (min_y + max_y) / 2.0f;

    vertices.clear();
    velocities.clear();
    std::srand((unsigned)std::time(nullptr));

    // 生成网格：i 控制 x 坐标，j 控制 y 坐标
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            float x = i * 4.0f/(n-1) - 2.0f + offsetX;
            float y = j * 4.0f/(n-1) - 2.0f + offsetY;
            // 生成噪声 ∈ [-0.05, +0.05]
            float noise = (((std::rand() % 1000) / 1000.0f) - 0.5f) * 0.1f;
            // 利用 y 坐标构造倒 V：中间（y=center_y）最高，两侧降低
            float z = baseline - amplitude * fabs(y - center_y) + noise;
            vertices.push_back(x);
            vertices.push_back(y);
            vertices.push_back(z);

            // 初始速度：采用 y 方向运动
            // 当 y < center_y 时（左侧）速度为正，且较大；y > center_y 时速度为负，较小；正中为 0
            float v_left  = 1.0f;  // 左侧最大速度
            float v_right = 0.5f;  // 右侧最大速度
            float vy;
            if (y < center_y) {
                vy = v_left * ((center_y - y) / (center_y - min_y));
            } else if (y > center_y) {
                vy = - v_right * ((y - center_y) / (max_y - center_y));
            } else {
                vy = 0.0f;
            }
            float vx = 0.0f;
            float vz = 0.0f;
            velocities.push_back(vx);
            velocities.push_back(vy);
            velocities.push_back(vz);
        }
    }

    // 分配 GPU 版 relax 操作的临时缓冲区 Temp，并置 0
    hipMalloc(&Temp, sizeof(float3)*n*n); CUERR
    hipMemset(Temp, 0, sizeof(float3)*n*n); CUERR

    // 根据布料网格计算约束参数
    // 这里以 x 坐标上相邻顶点距离为参考（均为 4/(n-1)）
    cnstr_two = vertices[3*n] - vertices[0];
    cnstr_dia = sqrt(2 * cnstr_two * cnstr_two);

    // 初始化法线数据
    normals.resize(3*n*n, 0.0f);
    update_normals((float3*) vertices.data(), (float3*) normals.data(), n);

    // 初始化颜色数据：全部设置为红色（RGBA:1,0,0,0.9）
    colors.clear();
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            colors.push_back(1.0f); // R
            colors.push_back(0.0f); // G
            colors.push_back(0.0f); // B
            colors.push_back(0.9f); // A
        }
    }

    // 构造索引数组，用于三角带绘制
    indices.clear();
    for (int i = 0; i < n-1; i++){
        int base = i * n;
        indices.push_back(base);
        for (int j = 0; j < n; j++){
            indices.push_back(base + j);
            indices.push_back(base + j + n);
        }
        indices.push_back(base + 2*n - 1);
    }

    // 创建并上传 OpenGL 缓冲区数据
    glGenBuffers(1, &vbo);
    glGenBuffers(1, &cbo);
    glGenBuffers(1, &nbo);
    glGenBuffers(1, &ibo);

    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glBufferData(GL_ARRAY_BUFFER, sizeof(GLfloat)*vertices.size(), vertices.data(), GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    hipGraphicsGLRegisterBuffer(&vbo_resource, vbo, cudaGraphicsMapFlagsWriteDiscard);

    glBindBuffer(GL_ARRAY_BUFFER, nbo);
    glBufferData(GL_ARRAY_BUFFER, sizeof(GLfloat)*normals.size(), normals.data(), GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    hipGraphicsGLRegisterBuffer(&vbo_resource2, nbo, cudaGraphicsMapFlagsWriteDiscard);

    glBindBuffer(GL_ARRAY_BUFFER, cbo);
    glBufferData(GL_ARRAY_BUFFER, sizeof(GLfloat)*colors.size(), colors.data(), GL_STATIC_DRAW);

    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, ibo);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(GLuint)*indices.size(), indices.data(), GL_STATIC_DRAW);

    // 分配设备侧速度数组 d_velocities，并将 host 侧数据复制过去
    int total = n * n;
    if (d_velocities) hipFree(d_velocities);
    hipMalloc((void**)&d_velocities, total * 3 * sizeof(float));
    hipMemcpy(d_velocities, velocities.data(), total * 3 * sizeof(float), hipMemcpyHostToDevice);

    // 分配设备侧锁数组 d_locks，初值置 0
    if (d_locks) hipFree(d_locks);
    hipMalloc((void**)&d_locks, total * sizeof(int));
    hipMemset(d_locks, 0, total * sizeof(int));

    start_time = std::chrono::high_resolution_clock::now();
}

/**************************************************
 * OpenGL 初始化函数
 **************************************************/
void init_GL(int *argc, char **argv) {
    std::srand((unsigned)std::time(nullptr));
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("Cloth Simulation: Main-View Inverted V with Left>Right Velocity (CUDA)");

    glutDisplayFunc(display);

    glewInit();

    glClearColor(1.0, 1.0, 1.0, 1.0);
    glEnable(GL_DEPTH_TEST);
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

    GLfloat mat_specular[] = {0.8f, 0.8f, 0.8f, 1.0f};
    GLfloat mat_shininess[] = {50.0f};
    GLfloat light_position[] = {1.0f, 1.0f, 1.0f, 0.0f};
    glShadeModel(GL_SMOOTH);
    glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
    glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
    glLightfv(GL_LIGHT0, GL_POSITION, light_position);

    glEnable(GL_COLOR_MATERIAL);
    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);

    glViewport(0, 0, window_width, window_height);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width/(GLfloat)window_height, 0.1, 10.0);
    glTranslatef(0.0, 0.0, -6.0);
    // 调整视角，使主视效果呈现倒 V 状态
    glRotated(300, 1, 0, 0);
    glRotated(270, 0, 0, 1);
}

/**************************************************
 * 主函数
 **************************************************/
int main(int argc, char **argv) {
    init_GL(&argc, argv);
    init_data(N);
    glutMainLoop();
    return 0;
}
